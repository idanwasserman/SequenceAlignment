#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"


// Error handling macro

#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        fprintf(stderr, "CUDA error calling \""#call"\", code is %d\n", err); \
        exit(EXIT_FAILURE); }
              


// Device helping unctions 


/*
CUDA_AreLettersExchangeable gets 2 characters (c and sub), and the conservative matrix
the function checks if the original character (c) is allowed to be substituted by another character (sub)
the substitute is allowed if there is no conservative group that contains both letters
*/
__device__ int CUDA_AreLettersExchangeable(char c, char sub, int* cons_mat)
{
	if(sub == HYPHEN)
		return 1;

	int pos1 = ABC_SIZE * (sub - 'A') + (c - 'A');
	int pos2 = ABC_SIZE * (c - 'A') + (sub - 'A');

	if(cons_mat[pos1] != 1 && cons_mat[pos2] != 1) // sub and c are not in the same conservative group
		return 1;
	else
		return 0;
}


/*
CUDA_IsBetterForGoal gets 2 numbers (d and best_d) and the goal of the program (MAX/MIN)
the function returns 1 if the number d is better for the program than the best_d
else - function returns 0
*/
__device__ int CUDA_IsBetterForGoal(double d, double best_d, int goal)
{
	if((goal == MAXIMUM && d > best_d) || (goal == MINIMUM && d < best_d))
		return 1;
	
	return 0;
}


/*
CUDA_GetLettersWeight gets 2 characters (c1 and c2), the conservative and semi conservative matrices and weights array
the function checks the relation between the 2 characters according to the matrices
and returns the matching weight for this pair of characters
*/
__device__ double CUDA_GetLettersWeight(char c1, char c2, int* cons_mat, int* semi_cons_mat, double* W)
{
	if(c1 == HYPHEN || c2 == HYPHEN)
	{
		if(c1 == c2)
			return W[STAR];
		else
			return -W[SPACE];
	}

	int pos1 = ABC_SIZE * (c1 - 'A') + (c2 - 'A');
	int pos2 = ABC_SIZE * (c2 - 'A') + (c1 - 'A');

	if(c1 == c2) // same letter
	{
		return W[STAR];
	}
	else
	{
		if(cons_mat[pos1] == 1 || cons_mat[pos2] == 1) // same conservative group
		{
			return -W[COLON];
		}
		else if(semi_cons_mat[pos1] == 1 || semi_cons_mat[pos2] == 1) // same semi-conservative group
		{
			return -W[POINT];
		}
		else
		{
			return -W[SPACE];
		}
	}
}


// Global functions

/*
this global function finds the best mutant for a specific offset between sequence #1 & sequnce #2
*/
__global__ void
	findBestMutant(const char* seq1, const char* seq2, char* best_seq, int length, int from, int best_offset, int* cons_mat, int* semi_cons_mat, double* W, int goal)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	
	// Check if current thread has a job to do
	if(index < length)
	{
		// Each thread take care of a different letter of sequence 2

		// current letters of seq1 & seq2 and best char for this pair
		char current_letter_seq1 = seq1[index + best_offset + from];
		char current_letter_seq2 = seq2[index];
		char best_char;

		// current & best weight for this letter in seq2
		double hyphen_weight = CUDA_GetLettersWeight(current_letter_seq1, HYPHEN, cons_mat, semi_cons_mat, W);
		double current_weight = CUDA_GetLettersWeight(current_letter_seq1, current_letter_seq2, cons_mat, semi_cons_mat, W);
		double best_weight;
		if(CUDA_IsBetterForGoal(current_weight, hyphen_weight, goal))
		{
			best_weight = current_weight;
			best_char = current_letter_seq2;
		}
		else
		{
			best_weight = hyphen_weight;
			best_char = HYPHEN;
		}
		
		for(char c = 'A'; c <= 'Z'; c++) // for each word in the ABC
		{
			if(c == current_letter_seq2)
				continue;

			if(CUDA_AreLettersExchangeable(current_letter_seq2, c, cons_mat))
			{
				// get current wight of letters between current_letter_seq1 & c
				current_weight = CUDA_GetLettersWeight(current_letter_seq1, c, cons_mat, semi_cons_mat, W);

				if(current_weight == best_weight)
					continue;

				if (CUDA_IsBetterForGoal(current_weight, best_weight, goal))
				{
					best_weight = current_weight;
					best_char = c;
				}
			}
		}

		best_seq[index] = best_char;
	}
}


/*
this global function fills the best-scores array
for each cell in the array - the function saves the best score that matches the goal of the program (MAX/MIN)
each cell represent a different offset between sequence #1 & sequence #2
*/
__global__  void
	fillScoreArr(const char* seq1, const char* seq2, int length, int from, int to, int* cons_mat, int* semi_cons_mat, double* W, double* best_score_arr, int goal)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
	int limit = to - from;

	// Check if current thread has a job to do
	if(index < limit)
	{	
		
		// Each thread take care of a different offset
		int my_offset = index + from;
		best_score_arr[index] = 0;

		for(int j = 0; j < length; j++) // for each letter in sequence #2
		{
			// current character of seq1 & seq2
			char current_letter_seq1 = seq1[my_offset + j];
			char current_letter_seq2 = seq2[j];

			// current & best weight for this round
			double hyphen_weight = CUDA_GetLettersWeight(current_letter_seq1, HYPHEN, cons_mat, semi_cons_mat, W);
			double current_weight = CUDA_GetLettersWeight(current_letter_seq1, current_letter_seq2, cons_mat, semi_cons_mat, W);
			double best_weight;
			if(CUDA_IsBetterForGoal(current_weight, hyphen_weight, goal))
				best_weight = current_weight;
			else
				best_weight = hyphen_weight;

			for(char c = 'A'; c <= 'Z'; c++) // for each word in the ABC
			{
				if(c == current_letter_seq2)
					continue;

				if(CUDA_AreLettersExchangeable(current_letter_seq2, c, cons_mat))
				{
					current_weight = CUDA_GetLettersWeight(current_letter_seq1, c, cons_mat, semi_cons_mat, W);
						
					if(current_weight == best_weight)
						continue;

					if (CUDA_IsBetterForGoal(current_weight, best_weight, goal))
					{
						best_weight = current_weight;
					}
				}
			}

			best_score_arr[index] += best_weight;
		}
	}
}


int computeOnGPU(const char* seq1, const char* seq2, char* best_seq, int len1, int len2, int from, int to, int* cons_mat, int* semi_cons_mat, double* W, double* best_score, int* best_offset, int goal) 
{
	int size = to - from; /* number of offsets to take care */

	
    // Allocate data on GPU memory
    char *d_seq1 = NULL, *d_seq2 = NULL, *d_best_seq = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_seq1, len1 * sizeof(char)));
    CUDA_CHECK(hipMalloc((void **)&d_seq2, len2 * sizeof(char)));
	CUDA_CHECK(hipMalloc((void **)&d_best_seq, len2 * sizeof(char)));
    
    int *d_cons_mat = NULL, *d_semi_cons_mat = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_cons_mat, ABC_SIZE * ABC_SIZE * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_semi_cons_mat, ABC_SIZE * ABC_SIZE * sizeof(int)));
    
    double *d_W = NULL, *d_best_score_arr = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_W, NUM_OF_WEIGHTS * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&d_best_score_arr, size * sizeof(double)));
    
    
	// Copy to GPU memory
    CUDA_CHECK(hipMemcpy(d_seq1, seq1, len1 * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_seq2, seq2, len2 * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_best_seq, seq2, len2 * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_cons_mat, cons_mat,  ABC_SIZE * ABC_SIZE * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_semi_cons_mat, semi_cons_mat,  ABC_SIZE * ABC_SIZE * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_W, W, NUM_OF_WEIGHTS * sizeof(double), hipMemcpyHostToDevice));
	
	
	// Fill best scores array
	// Run Kernel
    int threads_per_block = 256;
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;
    fillScoreArr<<<blocks_per_grid, threads_per_block>>>(d_seq1, d_seq2, len2, from, to, d_cons_mat, d_semi_cons_mat, d_W, d_best_score_arr, goal);
    CUDA_CHECK(hipGetLastError());

	    
	// Copy data back to CPU memory
	double* best_score_arr = (double*)malloc(size * sizeof(double));
    CUDA_CHECK(hipMemcpy(best_score_arr, d_best_score_arr, size * sizeof(double), hipMemcpyDeviceToHost));
    

	// Find best Offset in array using OpenMp
	omp_set_num_threads(4);
	*best_offset = 0;
	int i;
	*best_score = best_score_arr[0];
#pragma omp parallel for
	for(i = 1; i < size; i++)
	{
		if ((goal == MAXIMUM && *best_score < best_score_arr[i]) || (goal == MINIMUM && *best_score > best_score_arr[i]))
		{
#pragma omp critical
		{
			*best_offset = i;
			*best_score = best_score_arr[i];	
		}
		}
	}  
	

	// Find best mutant of best offset
	// Run Kernel
    threads_per_block = 256;
    blocks_per_grid = (len2 + threads_per_block - 1) / threads_per_block;
    findBestMutant<<<blocks_per_grid, threads_per_block>>>(d_seq1, d_seq2, d_best_seq, len2, from, *best_offset, d_cons_mat, d_semi_cons_mat, d_W, goal);
    CUDA_CHECK(hipGetLastError());


    // Copy data back to CPU memory
    CUDA_CHECK(hipMemcpy(best_seq, d_best_seq, len2 * sizeof(char), hipMemcpyDeviceToHost));


    // Free GPU memory
    CUDA_CHECK(hipFree(d_seq1));
    CUDA_CHECK(hipFree(d_seq2));
	CUDA_CHECK(hipFree(d_best_seq));
    CUDA_CHECK(hipFree(d_cons_mat));
    CUDA_CHECK(hipFree(d_semi_cons_mat));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_best_score_arr));

	free(best_score_arr);
    

    return 0;
}

